
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX_SECRET 200000
#define KEY_SIZE 8
#define BUFFER 512

__global__
void exor(const int size, const char *secret)
{
    char key[KEY_SIZE+1], temp[MAX_SECRET];

    key[0] = blockIdx.x/10 + 48;
    key[1] = blockIdx.x%10 + 48;
    key[2] = blockIdx.y/10 + 48;
    key[3] = blockIdx.y%10 + 48;
    key[4] = blockIdx.z + 48;
    key[5] = threadIdx.x + 48;
    key[6] = threadIdx.y + 48;
    key[7] = threadIdx.z + 48;

    for (int i = 0; i < size; i++)
    {
        temp[i] = secret[i] ^ key[i % KEY_SIZE];

        switch(temp[i])
        {
        case '|':
        case '~':
        case '^':
        case '*':
        case '+':
        case '_':
        case '{':
        case '}':
        case '\\':
        case '#':
            return;
        }
    }

    temp[size] = '\0';
    key[KEY_SIZE] = '\0';
    printf("Key: [%s]\n%s\n\n", key, temp);
}

int
main(int argc, char *argv[])
{
    hipError_t err = hipSuccess;

    if(argc < 2)
    {
        fprintf(stderr, "No imput file specified!\n");
        exit(EXIT_FAILURE);
    }

    FILE *f = fopen(argv[1], "r");

    if(f == NULL)
    {
        fprintf(stderr, "Failed to open imput file!\n");
        exit(EXIT_FAILURE);
    }

    int n;
    char secret[MAX_SECRET];
    char *p = secret;

    while (n = fread((void *) p, 1, (p - secret + BUFFER < MAX_SECRET) ? BUFFER : secret + MAX_SECRET - p, f))
        p += n;

    fclose(f);
    int size = p - secret;
    secret[size] = '\0';

    char *d_secret = NULL;
    err = hipMalloc((void **)&d_secret, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate d_secret! (error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_secret, secret, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy from host to device! (error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 blocksPerGrid(100, 100, 10);
    dim3 threadsPerBlock(10, 10, 10);

    exor<<<blocksPerGrid, threadsPerBlock>>>(size, d_secret);

    if (hipGetLastError() != hipSuccess)
    {
        fprintf(stderr, "Failed to launch exor kernel!\n");
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_secret);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_secret! (error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (hipDeviceReset() != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device!\n");
        exit(EXIT_FAILURE);
    }

    fprintf(stderr, "Done\n");
    return 0;
}
